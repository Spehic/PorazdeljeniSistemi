
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif

__global__ void hello(char *message) {
    printf("%s\n", message);
}

__global__ void process(unsigned char *img_in, unsigned char *img_out, int width, int height) {
    // row
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    // col
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int ipx = i * width + j;

    unsigned char medArr[9];
    while (ipx < width * height) {
        
        //i am on the left wall
        if( ipx % width == 0 ) {
            // if i am on top left
            if ( ipx - width < 0 ) medArr[0] = img_in[ipx];
            else medArr[0] = img_in[ ipx - width ];

            medArr[3] = img_in[ipx];

            //if a am on bottom left
            if ( ipx + width >= width * height ) medArr[6] = img_in[ipx];
            else medArr[6] = img_in[ ipx + width ];
        }else {
			//if i am on top
			if ( ipx < width ) 
				medArr[0] = img_in[ ipx - 1 ];
			 else 
				medArr[0] = img_in[ ipx - width - 1 ];
			
            
            medArr[3] = img_in[ ipx - 1 ];

			//if i am on bottom
			if ( ipx + width >= width * height ) {
				medArr[6] = img_in[ ipx - 1];
			} else {
				medArr[6] = img_in[ ipx + width - 1];
			}
        }

         //i am on the right wall
        if( ipx % width == width - 1 ) {
            // if i am on top right
            if ( ipx - width < 0 ) medArr[2] = img_in[ipx];
            else medArr[2] = img_in[ ipx - width ];

            medArr[5] = img_in[ipx];

            //if a am on bottom right
            if ( ipx + width > width * height ) medArr[8] = img_in[ipx];
            else medArr[8] = img_in[ ipx + width ];
        }else {
			//if i am on top
			if (ipx < width) {
				medArr[2] = img_in[ ipx + 1 ];
			} else { 
				medArr[2] = img_in[ ipx - width + 1 ];
			}
            
            medArr[5] = img_in[ ipx + 1 ];

			//if i am on bottom
			if (ipx + width >= width * height) {
				medArr[8] = img_in[ ipx + 1];
			} else {
				medArr[8] = img_in[ ipx + width + 1];
			}
        }

        // i am on top
        if( ipx < width ) {
            // if i am on top left
            if ( ipx - 1 < 0 ) medArr[0] = img_in[ipx];
            else medArr[0] = img_in[ ipx - 1 ];

            medArr[1] = img_in[ipx];

            //if a am on top right
            if ( (ipx + 1) % width == 0 ) medArr[2] = img_in[ipx];
            else medArr[2] = img_in[ ipx + 1 ];
        }else {
			//if i am on left wall
			if (ipx % width == 0) {
				medArr[0] = img_in[ ipx - width ];
			} else {
				medArr[0] = img_in[ ipx - width - 1 ];
			}
            
            medArr[1] = img_in[ ipx - width ];

			//if i am on right wall
			if  ( (ipx + 1) % width == 0 )  {
				medArr[2] = img_in[ ipx - width ];
			} else {
            	medArr[2] = img_in[ ipx - width + 1];
			}
        }


        // i am on bottom
        if( ipx + width >= width * height ) {
            // if i am on bottom left
            if ( ipx % width == 0 ) medArr[6] = img_in[ipx];
            else medArr[6] = img_in[ ipx - 1 ];

            medArr[7] = img_in[ipx];

            //if a am on bottom right
            if ( ipx + 1 >= width * height ) medArr[8] = img_in[ipx];
            else medArr[8] = img_in[ ipx + 1 ];
        }else {
			//if i am on left wall
			if (ipx % width == 0) {
				medArr[6] = img_in[ ipx + width ];
			} else {
				medArr[6] = img_in[ ipx + width - 1 ];
			}
            
            medArr[7] = img_in[ ipx + width ];

			//if i am on right wall
			if  ((ipx + 1) % width == 0 ) { 
				medArr[8] = img_in[ ipx + width ];
			} else {
				medArr[8] = img_in[ ipx + width + 1 ];
			}
        }

        medArr[4] = img_in[ ipx ];
        
        for (int a = 0; a < 8; a++) {
            for (int b = 0; b < 9 - a - 1; b++) {
                if (medArr[b] > medArr[b + 1]) {
                    int temp = medArr[b];
                    medArr[b] = medArr[b + 1];
                    medArr[b + 1] = temp;
                }
            }
        }


        img_out[ipx] = medArr[4];
        ipx += blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    }
}



#ifdef __cplusplus
}
#endif